#include "sumColsMatrix.cuh"
#include "sumColsMatrix.hpp"
#include <stdio.h>

#define ROW 1024
#define COL 512


int main(int argc, char** argv){

  int N, M;

  /*if (argc == 3){ //case input order: row col
    N = atoi(argv[1]);
    M = atoi(argv[2]);
  }
  else if (argc == 2){ //case number cols = number rows
    N = N = atoi(argv[1]);
  }
  else if (argc > 3){ //error case
    printf("Error typing dimension of a matrix (rows,columns), entered more than two values\n");
    return -1;
  }
  else{ //run example with predefined size 1024x512
    N = ROW;
    M = COL;
  }*/

  N = ROW;
  M = COL;


  /*int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Shared Memory per block: %d\n",
           prop.sharedMemPerBlock);
    printf("  Dimension of a grid: (%d,%d,%d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("  Warp size: %d\n",
           prop.warpSize);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }*/

  int *Mh;
  int *Nh;

  Mh = new int[N*M];

  for (int i = 0; i < N; ++i)
    for (int j = 0; j < M; ++j)
      Mh[i*M + j] = i;

  Nh = new int[M];

  sumColsMatrixGPU(Mh,Nh,M,N);

  for (int i = 0; i < M; ++i)
    printf("%d ",Nh[i]);
  printf("\n");

  delete Mh;
  delete Nh;

  return 0;
}
