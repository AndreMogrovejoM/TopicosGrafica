#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <cmath>

#ifdef __HIPCC__

#define SIZE_TRANS 6

__constant__ float M[SIZE_TRANS];

__global__
void affin_transform_kernel(uchar3* in_img, uchar3* out_img, size_t width, size_t height)
{
  uint tid = blockIdx.x*blockDim.x + threadIdx.x;
  uint x = tid/width;
  uint y = tid%width;

  uint nx,ny;

  nx = M[0]*x + M[1]*y + M[2];
  ny = M[3]*x + M[4]*y + M[5];

  uint ntid = nx*width + ny;
  out_img[ntid] = in_img[tid];
}

void affin_transform(uchar3* d_in_img, uchar3* d_out_img, float* trans, size_t width, size_t height)
{
  hipMemcpyToSymbol(HIP_SYMBOL(M), trans, SIZE_TRANS*sizeof(float));

  dim3 gridDim(std::ceil(width*height/warpSize),1,1);
  dim3 blockDim(warpSize,1,1);
  affin_transform_kernel<<<gridDim,blockDim>>>(d_in_img,d_out_img,width,height);
}

#endif