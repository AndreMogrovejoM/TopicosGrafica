
#include "hip/hip_runtime.h"

#include <math.h> 
#include <stdio.h>


#define ThreadsPerBlock 512


__global__
void GammaKernel(const uchar4* const rgbaImage, uchar4* const outputImage, int numRows, int numCols, float gamma)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < numRows * numCols) {
		uchar4 px = rgbaImage[i]; 
		unsigned char rcolor = round(pow((px.x / 255.0f), (1.0f / gamma)) * 255.0f);
		outputImage[i].x = (rcolor > 255.0f) ? 255.0f : rcolor;
		unsigned char gcolor = round(pow((px.y / 255.0f), (1.0f / gamma)) * 255.0f);
		outputImage[i].y = (gcolor > 255.0f) ? 255.0f : gcolor;
		unsigned char bcolor = round(pow((px.z / 255.0f), (1.0f / gamma)) * 255.0f);
		outputImage[i].z = (bcolor > 255.0f) ? 255.0f : bcolor;

		
	}

}

void CorreccionGamma(uchar4* const d_rgbaImage, uchar4* const d_outputImage, size_t numRows, size_t numCols, float gamma)
{
	long long int total_px = numRows * numCols;  
	long int grids_n = ceil(total_px / ThreadsPerBlock); 
	const dim3 blockSize(ThreadsPerBlock, 1, 1);
	const dim3 gridSize(grids_n, 1, 1);
	GammaKernel <<<gridSize, blockSize >>> (d_rgbaImage, d_outputImage, numRows, numCols, gamma);
	hipDeviceSynchronize();
}