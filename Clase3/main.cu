//#include "shared.cuh"
#include <stdio.h>

#define ROW 1024
#define COL 512


int main(int argc, char** argv){
  int row,col;

  if (argc == 3){ //case input order: row col
    row = atoi(argv[1]);
    col = atoi(argv[2]);
  }
  else if (argc == 2){ //case number cols = number rows
    row = col = atoi(argv[1]);
  }
  else if (argc > 3){ //error case
    printf("Error typing dimension of a matrix (rows,columns), entered more than two values\n");
    return -1;
  }
  else{ //run example with predefined size 1024x512
    row = ROW;
    col = COL;
  }

  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Shared Memory per block: %d\n",
           prop.sharedMemPerBlock);
    printf("  Registers per block: %d\n",
           prop.regsPerBlock);
    printf("  Warp size: %d\n",
           prop.warpSize);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
  return 0;
}
