#include "hip/hip_runtime.h"
#include "utils.h"

__global__
void image_sum_kernel(const uchar3* const inImg1, const uchar3* const inImg2, uchar3* const outImg, size_t n)
{
  size_t id = blockIdx.x*blockDim.x + threadIdx.x; 
  if (id < n){
    uchar3 px_img1 = inImg1[id];
    uchar3 px_img2 = inImg2[id];

    outImg[id].x = px_img1.x + px_img2.x;
    outImg[id].y = px_img1.y + px_img2.y;
    outImg[id].z = px_img1.z + px_img2.z;
  }
}

void image_sum(uchar3* const d_inImg1, uchar3* const d_inImg2, uchar3* const d_outImg, size_t n)
{
  const dim3 gridSize(n/warpSize,1,1);
  const dim3 blockSize(warpSize,1,1);
  sum_kernel<<<gridSize,blockSize>>>(d_inImg1, d_inImg2, d_outImg, n);
}
