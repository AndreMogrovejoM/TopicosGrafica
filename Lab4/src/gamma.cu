#include "hip/hip_runtime.h"
#include <cmath>
#include "utils.h"

__global__
void gamma_kernel(const uchar3* const inImg, uchar3* const outImg, const size_t n, const int gamma)
{
  size_t id = blockIdx.x*blockDim.x + threadIdx.x; 
  if (id < n){
    uchar3 px_img = inImg[id];

    outImg[id].x = 255 * pow(px_img.x / 255, gamma);
    outImg[id].y = 255 * pow(px_img.y / 255, gamma);
    outImg[id].z = 255 * pow(px_img.z / 255, gamma);
  }
}

void gamma_func(uchar3* const d_inImg, uchar3* const d_outImg, size_t &n, int gamma)
{
  const dim3 gridSize(n/warpSize,1,1);
  const dim3 blockSize(warpSize,1,1);
  gamma_kernel<<<gridSize,blockSize>>>(d_inImg, d_outImg, n, 1/gamma);
}