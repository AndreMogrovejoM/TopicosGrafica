#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <iostream>  
#include <fstream>   
#include <iomanip>   
#include <windows.h>
#include <io.h>                  
#include <stdio.h>
#include<conio.h>
#include <cstdlib>
#include "cstdlib"
#include <process.h>
#include <stdlib.h>
#include <malloc.h>
#include <ctime>

//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

using namespace std;

using namespace std;

#define BLOCK_WIDTH 16 
#define BLOCK_HEIGHT 16

/*******************/
/* Div_entre_cero FUNCTION */
/*******************/
int Div_entre_cero(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__global__ void Conv_2d_kernel_avg_pool(unsigned short* Img_entrada, unsigned short* Img_salida, int Img_w, int Img_h) {

    __shared__ unsigned short surround[BLOCK_WIDTH * BLOCK_HEIGHT][9];

    int iterador;
    //Ids de threads y blocks
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;

    //Comprobando que el kernel este dentro de la imagen
    if ((x >= (Img_w - 1)) || (y >= Img_h - 1) || (x == 0) || (y == 0)) return;

    // --- LLenando memoria compartida
    iterador = 0;
    for (int r = x - 1; r <= x + 1; r++) {
        for (int c = y - 1; c <= y + 1; c++) {
            surround[tid][iterador] = Img_entrada[c * Img_w + r];
            iterador++;
        }
    }

    // Ordenamos la memoria compartida con bubblesort para calcular la media
    for (int i = 0; i < 5; ++i) {

        // Obtenemos el menor valor
        int minval = i;
        for (int l = i + 1; l < 9; ++l)
            if (surround[tid][l] < surround[tid][minval]) minval = l;

        // Ponemos el menor valor en su lugar
        unsigned short temp = surround[tid][i];
        surround[tid][i] = surround[tid][minval];
        surround[tid][minval] = temp;
    }

    // Elegimos el valor medio
    Img_salida[(y * Img_w) + x] = surround[tid][4];

    __syncthreads();

}


int main()
{
    //Dimension de la imagen
    const int Img_w = 1580;
    const int Img_h = 1348;

    // Abriendo imagen en formato raw
    ifstream is; is.open("ps5.raw", ios::binary);

    // Longitud del archivo
    is.seekg(0, ios::end);
    int img_long = is.tellg();
    is.seekg(0, ios::beg);

    // Lectura de la imagen y almacenaje en host
    unsigned short* Img_entrada_host = new unsigned short[img_long * sizeof(char) / sizeof(unsigned short)];
    is.read((char*)Img_entrada_host, img_long);
    is.close();

    // Comprobacion de error al separar memoria
    unsigned short* forFirstCudaMalloc; gpuErrchk(hipMalloc((void**)&forFirstCudaMalloc, img_long * sizeof(unsigned short)));
    gpuErrchk(hipFree(forFirstCudaMalloc));

    //Separamos memoria en host y device
    unsigned short* Img_salida_host = (unsigned short*)malloc(img_long);
    unsigned short* Img_entrada; gpuErrchk(hipMalloc((void**)&Img_entrada, img_long * sizeof(unsigned short)));
    unsigned short* Img_salida; gpuErrchk(hipMalloc((void**)&Img_salida, img_long * sizeof(unsigned short)));

    // Copiando la data del host al device
    gpuErrchk(hipMemcpy(Img_entrada, Img_entrada_host, img_long, hipMemcpyHostToDevice));// copying Host Data To Device Memory For Filtering

    // Calculo de tamaño grids y blocks
    const dim3 grid(Div_entre_cero(Img_w, BLOCK_WIDTH), Div_entre_cero(Img_h, BLOCK_HEIGHT), 1);
    const dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT, 1);

    //Variables para la medicion de tiempo
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Ejecucion del kernel
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(Conv_2d_kernel_avg_pool), hipFuncCachePreferShared);
    Conv_2d_kernel_avg_pool << <grid, block >> > (Img_entrada, Img_salida, Img_w, Img_h);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    //Imprimir tiempos de ejecución en ms
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Ejecución completa - time:  %3.3f ms \n", time);

    //Moviendo resultados del device al host
    gpuErrchk(hipMemcpy(Img_salida_host, Img_salida, img_long, hipMemcpyDeviceToHost));

    // Escribiendo resultados en archivo de salida .raw
    ofstream of2;     of2.open("ps5_filtrada.raw", ios::binary);
    of2.write((char*)Img_salida_host, img_long);
    of2.close();

    cout << "\n Press any Key!!";
    //Liberando memoria del device
    gpuErrchk(hipFree(Img_entrada));
    //Liberando memoria del host
    delete Img_entrada_host;
    delete Img_salida_host;

    return 0;
}
